#include "hip/hip_runtime.h"
/* sha.cu
 * ECE-565, Project 3
 * Connor Horne
 * Cuda routine for sha1 cracking
 */

#include "sha.h"
#include <stdio.h>

// launch the kernel
//@
//@
//@
void launch(word *hash_tmp, unsigned char *result);

// kernel entrence to brute force sha test
// @ device_result is result from the SHA crack
// @ device_hash is the input hash
__global__ void smash(volatile unsigned char *device_result,
                      unsigned int *device_hash);

// prepares W for SHA1
// @ W is 80 int array for sha calculation
// @ test word is the ascii array of the test word
// @ length is the length of the indies that are being tested
__device__ void memInit(unsigned int *W, unsigned char *test_word, int length);

// initialize word to global mem
// @test is 8 unsigned char array to hold test word vals
__device__ int initTestWord(unsigned char *test);

// shift the word by block*dim
// @test is 8 unsigned test word
// @loc is the global location of the thread
__device__ int shift(unsigned char *test, unsigned long long *loc);

// Main loop SHA logical functions f1 to f4
__device__ inline word f1(word x, word y, word z) {
  return ((x & y) | (~x & z));
}
__device__ inline word f2(word x, word y, word z) { return (x ^ y ^ z); }
__device__ inline word f3(word x, word y, word z) {
  return ((x & y) | (x & z) | (y & z));
}
__device__ inline word f4(word x, word y, word z) { return (x ^ y ^ z); }

// SHA init constants
#define I1 1732584193U
#define I2 4023233417U
#define I3 2562383102U
#define I4 271733878U
#define I5 3285377520U

// 32-bit rotate
__device__ inline word ROT(word x, int n) {
  return ((x << n) | (x >> (32 - n)));
}

// calculation functions for 80 rounds of SHA1
#define CALC1(i)                                                               \
  temp = ROT(A, 5) + f1(B, C, D) + W[i] + E + 1518500249U;                     \
  E = D;                                                                       \
  D = C;                                                                       \
  C = ROT(B, 30);                                                              \
  B = A;                                                                       \
  A = temp

#define CALC2(i)                                                               \
  temp = ROT(A, 5) + f2(B, C, D) + W[i] + E + 1859775393U;                     \
  E = D;                                                                       \
  D = C;                                                                       \
  C = ROT(B, 30);                                                              \
  B = A;                                                                       \
  A = temp

#define CALC3(i)                                                               \
  temp = ROT(A, 5) + f3(B, C, D) + W[i] + E + 2400959708U;                     \
  E = D;                                                                       \
  D = C;                                                                       \
  C = ROT(B, 30);                                                              \
  B = A;                                                                       \
  A = temp

#define CALC4(i)                                                               \
  temp = ROT(A, 5) + f4(B, C, D) + W[i] + E + 3395469782U;                     \
  E = D;                                                                       \
  D = C;                                                                       \
  C = ROT(B, 30);                                                              \
  B = A;                                                                       \
  A = temp

// ascii constants constants
#define HIGH 126
#define LOW 32
#define BASE 95

// set the max search depth
#define MAX 6

// Offsets for 95^x
#define OFFSET1 95LL
#define OFFSET2 9120LL
#define OFFSET3 866495LL
#define OFFSET4 82317120LL
#define OFFSET5 7820126495LL
#define OFFSET6 742912017120LL
#define OFFSET7 70576641626495LL
#define OFFSET8 6704780954517120LL

// Launch Kernel Code
void launch(word *input_hash, unsigned char *result) {

  // device result is the found hash from the kernel run, cuda memory
  // device hash is input hash
  unsigned char *device_result;
  word *device_hash;

  hipMalloc((void **)&device_result, 10 * sizeof(unsigned char));
  hipMalloc((void **)&device_hash, 5 * sizeof(word));

  hipMemcpy(device_hash, input_hash, 5 * sizeof(word), hipMemcpyHostToDevice);
  hipMemset(device_result, 0, 10 * sizeof(unsigned char));

  // cuda timing of kernel
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  // call the kenel for searching for values

  // smash<<<1, 32>>>(max, device_result, device_hash);
  // smash<<<14,192>>>(max, device_result, device_hash);
  // smash<<<14, 1024>>>(device_result, device_hash);

  // 40 warps
  // this warp is faster than 100% occupency
  // smash<<<14*5, 256>>>(device_result, device_hash);

  // this with reg limit of 32 achieves 100% occupency with
  // -maxregcount 32
  smash<<<14 * 16, 128>>>(device_result, device_hash);

  hipEventRecord(stop);

  hipEventSynchronize(stop);

  // get possibly found result back from kernel
  hipMemcpy(result, device_result, 10 * sizeof(unsigned char),
             hipMemcpyDeviceToHost);

  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
    printf("Cuda error: %s.\n", hipGetErrorString(err));

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("kernel time: %.1f ms\n", milliseconds);

  // free mem
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(device_hash);
  hipFree(device_result);
}

// call function from non-cuda code
double crack(unsigned int *hash, unsigned char *result) {

  double time = 0;

  // time the entire kernel launch process
  clock_t test = clock();

  // call wrapper of kernel lanch
  launch(hash, result);

  time = ((double)clock() - test) / CLOCKS_PER_SEC;

  return time;
}

__global__ void smash(volatile unsigned char *result, word *hash) {

  int gtid = (blockDim.x * blockIdx.x) + threadIdx.x;

  // need to make sure this is placed in registers
  // unsigned char test_word[MAX];
  unsigned char test_word[8];

  word h0, h1, h2, h3, h4;

  // load input hash into local var
  h0 = hash[0];
  h1 = hash[1];
  h2 = hash[2];
  h3 = hash[3];
  h4 = hash[4];

  // initil test word
  // force word into registers
  test_word[0] = 0;
  test_word[1] = 0;
  test_word[2] = 0;
  test_word[3] = 0;
  test_word[4] = 0;
  test_word[5] = 0;
  test_word[6] = 0;
  test_word[7] = 0;

  // sets the test word to the gtid
  int length = initTestWord(test_word);

  // vars for SHA1 calc
  word W[80], A, B, C, D, E, temp;

  // init search position
  unsigned long long loc = gtid;

  while ((result[0] == 0) && (length < 7)) {

    // convert the test word to proper format for SHA
    // places bit representation in steps of 8
    // appends 8 to end and length to end
    memInit(W, test_word, length);

    // calculate sha1
    // unroll this loop to make sure W is placed in registers
#pragma unroll
    for (int i = 16; i < 80; i++) {
      W[i] = ROT((W[i - 3] ^ W[i - 8] ^ W[i - 14] ^ W[i - 16]), 1);
    }

    // Perform sha calculation
    A = I1;
    B = I2;
    C = I3;
    D = I4;
    E = I5;

    // 80 rounds
    CALC1(0);
    CALC1(1);
    CALC1(2);
    CALC1(3);
    CALC1(4);
    CALC1(5);
    CALC1(6);
    CALC1(7);
    CALC1(8);
    CALC1(9);
    CALC1(10);
    CALC1(11);
    CALC1(12);
    CALC1(13);
    CALC1(14);
    CALC1(15);
    CALC1(16);
    CALC1(17);
    CALC1(18);
    CALC1(19);
    CALC2(20);
    CALC2(21);
    CALC2(22);
    CALC2(23);
    CALC2(24);
    CALC2(25);
    CALC2(26);
    CALC2(27);
    CALC2(28);
    CALC2(29);
    CALC2(30);
    CALC2(31);
    CALC2(32);
    CALC2(33);
    CALC2(34);
    CALC2(35);
    CALC2(36);
    CALC2(37);
    CALC2(38);
    CALC2(39);
    CALC3(40);
    CALC3(41);
    CALC3(42);
    CALC3(43);
    CALC3(44);
    CALC3(45);
    CALC3(46);
    CALC3(47);
    CALC3(48);
    CALC3(49);
    CALC3(50);
    CALC3(51);
    CALC3(52);
    CALC3(53);
    CALC3(54);
    CALC3(55);
    CALC3(56);
    CALC3(57);
    CALC3(58);
    CALC3(59);
    CALC4(60);
    CALC4(61);
    CALC4(62);
    CALC4(63);
    CALC4(64);
    CALC4(65);
    CALC4(66);
    CALC4(67);
    CALC4(68);
    CALC4(69);
    CALC4(70);
    CALC4(71);
    CALC4(72);
    CALC4(73);
    CALC4(74);
    CALC4(75);
    CALC4(76);
    CALC4(77);
    CALC4(78);
    CALC4(79);

    A += I1;
    B += I2;
    C += I3;
    D += I4;
    E += I5;

    // check if the sha generated from test is equal to input sha
    // if true fill results buffer
    if (A == h0 && B == h1 && C == h2 && D == h3 && E == h4) {
      result[0] = test_word[0];
      result[1] = test_word[1];
      result[2] = test_word[2];
      result[3] = test_word[3];
      result[4] = test_word[4];
      result[5] = test_word[5];
      result[6] = test_word[6];
      result[7] = test_word[7];
    }

    // shift the word by a stride length of block*grid
    length = shift(test_word, &loc);
  }
  return;
}

/*
 * device function __device__ void memInit(uint, uchar, int)
 *
 * Prepare word for sha-1 (expand, add length etc)
 */
// could make various length based template versions
// can then unroll first for loop
__device__ void memInit(word *tmp, unsigned char input[], int length) {

// zero W array
// unroll it for placement in registers
#pragma unroll
  for (int i = 0; i < 80; i++) {
    tmp[i] = 0;
  }

  // switch statement
  // necessary in this single kernel launch
  // will result in input and W being placed in registers
  // in general will take words up to length and logical
  // or them into the word array's index (want chars as bits)
  // then append hex 80 to the last position after chars
  switch (length) {
  case 1:
    tmp[0] |= input[0] << 24;
    tmp[0] |= 128 << 16;

    break;
  case 2:
    tmp[0] |= input[0] << 24;
    tmp[0] |= input[1] << 16;
    tmp[0] |= 128 << 8;

    break;
  case 3:
    tmp[0] |= input[0] << 24;
    tmp[0] |= input[1] << 16;
    tmp[0] |= input[2] << 8;
    tmp[0] |= 128;

    break;
  case 4:
    tmp[0] |= input[0] << 24;
    tmp[0] |= input[1] << 16;
    tmp[0] |= input[2] << 8;
    tmp[0] |= input[3];
    tmp[1] |= (unsigned int)128 << 24;

    break;
  case 5:
    tmp[0] |= input[0] << 24;
    tmp[0] |= input[1] << 16;
    tmp[0] |= input[2] << 8;
    tmp[0] |= input[3];
    tmp[1] |= input[4] << 24;
    tmp[1] |= 128 << 16;

    break;
  case 6:
    tmp[0] |= input[0] << 24;
    tmp[0] |= input[1] << 16;
    tmp[0] |= input[2] << 8;
    tmp[0] |= input[3];
    tmp[1] |= input[4] << 24;
    tmp[1] |= input[5] << 16;
    tmp[1] |= 128 << 8;

    break;
  case 7:
    tmp[0] |= input[0] << 24;
    tmp[0] |= input[1] << 16;
    tmp[0] |= input[2] << 8;
    tmp[0] |= input[3];
    tmp[1] |= input[4] << 24;
    tmp[1] |= input[5] << 16;
    tmp[1] |= input[6] << 8;
    tmp[1] |= (unsigned int)128 << 24;

    break;
  }

  // Add length to end
  tmp[15] |= length * 8;
}

/*
 *
 *
 */
__device__ int initTestWord(unsigned char *test_word) {

  int gtid = (blockDim.x * blockIdx.x) + threadIdx.x;

  int length = 0;
  unsigned long long temp = gtid;

  // as there is a single kernel it can be offset by the previous search
  // one space goes from 0-94 and two spaces goes from 0-9025 but as its
  // running off of a striding global var then its from 95-9120 and
  // so on
  // this could be mitigated with muliple kernels and would probably
  // decrease the register count
  unsigned long long offset1 = 95;
  unsigned long long offset2 = (95 * 95) + 95;
  unsigned long long offset3 = (95 * 95 * 95) + (95 * 95) + 95;
  unsigned long long offset4 =
      (95 * 95 * 95 * 95) + (95 * 95 * 95) + (95 * 95) + 95;
  unsigned long long offset5 = (95LL * 95 * 95 * 95 * 95) +
                               (95LL * 95 * 95 * 95) + (95LL * 95 * 95) +
                               (95LL * 95) + 95LL;
  unsigned long long offset6 =
      (95LL * 95 * 95 * 95 * 95 * 95) + (95LL * 95 * 95 * 95 * 95) +
      (95LL * 95 * 95 * 95) + (95LL * 95 * 95) + (95LL * 95) + 95LL;

  // check if if num is above offset, if it is remove offset and
  // set all indicies to 32
  // this setup is designed to get compiler to place in registers
  // should be able to remove the first couple if statments as there is no
  // way that it could equal it here
  // note that 32 is the ascii offset
  if ((temp / offset6) > 0) {
    temp = temp - offset6;
    length = 7;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
    test_word[4] = 32;
    test_word[5] = 32;
    test_word[6] = 32;
  } else if ((temp / offset5) > 0) {
    temp = temp - offset5;
    length = 6;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
    test_word[4] = 32;
    test_word[5] = 32;
  } else if ((temp / offset4) > 0) {
    temp = temp - offset4;
    length = 5;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
    test_word[4] = 32;

  } else if ((temp / offset3) > 0) {
    temp = temp - offset3;
    length = 4;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;

  } else if ((temp / offset2) > 0) {
    temp = temp - offset2;
    length = 3;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;

  } else if ((temp / offset1) > 0) {
    temp = temp - offset1;
    length = 2;

    test_word[0] = 32;
    test_word[1] = 32;

  } else {
    length = 1;

    test_word[0] = 32;
  }

// perform a base conversion with 95
// as the result will be 0 if nothing present, can
// safetly unroll and use as dummy result for
// nesseacery place word in registers
#pragma unroll
  for (int i = 0; i < 8; i++) {
    test_word[i] += (unsigned char)(temp % BASE);
    temp /= BASE;
  }

  return length;
}

// increment word by stride length
__device__ int shift(unsigned char *test_word, unsigned long long *loc) {

  // get stride length and increment the global location
  int stride = (blockDim.x * gridDim.x);
  (*loc) += (unsigned long long)stride;

  unsigned long long temp = *loc;

  int length = 0;

  // see initword function for details
  unsigned long long offset1 = 95;
  unsigned long long offset2 = (95 * 95) + 95;
  unsigned long long offset3 = (95 * 95 * 95) + (95 * 95) + 95;
  unsigned long long offset4 =
      (95 * 95 * 95 * 95) + (95 * 95 * 95) + (95 * 95) + 95;
  unsigned long long offset5 = (95LL * 95 * 95 * 95 * 95) +
                               (95LL * 95 * 95 * 95) + (95LL * 95 * 95) +
                               (95LL * 95) + 95LL;
  unsigned long long offset6 =
      (95LL * 95 * 95 * 95 * 95 * 95) + (95LL * 95 * 95 * 95 * 95) +
      (95LL * 95 * 95 * 95) + (95LL * 95 * 95) + (95LL * 95) + 95LL;

  // see initword function for more details
  // remove offset, set length, set words to ascii offset
  if ((temp / offset6) > 0) {
    temp = temp - offset6;
    length = 7;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
    test_word[4] = 32;
    test_word[5] = 32;
    test_word[6] = 32;

  } else if ((temp / offset5) > 0) {
    temp = temp - offset5;
    length = 6;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
    test_word[4] = 32;
    test_word[5] = 32;

  } else if ((temp / offset4) > 0) {
    temp = temp - offset4;
    length = 5;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
    test_word[4] = 32;
  } else if ((temp / offset3) > 0) {
    temp = temp - offset3;
    length = 4;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;
    test_word[3] = 32;
  } else if ((temp / offset2) > 0) {
    temp = temp - offset2;
    length = 3;

    test_word[0] = 32;
    test_word[1] = 32;
    test_word[2] = 32;

  } else if ((temp / offset1) > 0) {
    temp = temp - offset1;
    length = 2;

    test_word[0] = 32;
    test_word[1] = 32;
  } else {
    length = 1;

    test_word[0] = 32;
  }

// ensure that compiler places in registers
// if result is greater, will be dummy result
// and will not harm the word
#pragma unroll
  for (int i = 0; i < 8; i++) {
    test_word[i] += (unsigned char)(temp % BASE);
    temp /= BASE;
  }

  return length;
}
